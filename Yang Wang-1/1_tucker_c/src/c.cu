/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.

 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 * Yang Wang
 * Depatment of ECE
 * University of Toronto
 */
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>


__global__ void ADD(float * A, float*O,int N,float x)
{
	int i = (blockDim.x * blockIdx.x + threadIdx.x) * N;
	if (i < N * 1024 * 1024)
	{
	O[i] = A[i]+x;
	}
}

int main(void)
{

	printf("%s\n%s\n%s\n","FIRSTNAME: Yang","LASTNAME: Wang","E-MAIL: tucker.wang@mail.utoronto.ca");
	printf("%-15s%-15s%-15s%-15s\n","N","CPUtoGPU(ms)","Kernel(ms)","GPUtoCPU(ms)");

	float x = 10.0;

	for(int N = 1; N <= 32; N++)
	{
	size_t size = N * 1024 * 1024 * sizeof(float);
	float * hA = (float *)malloc(size);
	float * hO = (float *)malloc(size);

	float * dA;
	hipMalloc((void**)&dA, size);
	float * dO;
	hipMalloc((void**)&dO, size);

	hipEvent_t end,end_1,end_2;
	hipEvent_t start,start_1,start_2;
	hipEventCreate(&start);
	hipEventCreate(&end);
	hipEventCreate(&start_1);
	hipEventCreate(&end_1);
	hipEventCreate(&start_2);
	hipEventCreate(&end_2);

	for( int i = 0; i< N * 1024 * 1024; i++)
	{
		hA[i] = rand()/(float)RAND_MAX;
	}

	hipEventRecord(start);
	hipMemcpy(dA,hA,size,hipMemcpyHostToDevice);
	hipEventRecord(end);
	hipEventSynchronize(end);
	float eTime = 0;
	hipEventElapsedTime(&eTime, start, end);

	int threadsPerBlock = 256;
	int blocksPerGrid = (N * 1024 * 1024 + threadsPerBlock - 1) / threadsPerBlock;

	hipEventRecord(start_1);
	ADD<<<blocksPerGrid,threadsPerBlock>>>(dA,dO,N,x);
	hipEventRecord(end_1);
	hipDeviceSynchronize ();
	hipEventSynchronize(end_1);
	float eTime1 = 0;
	hipEventElapsedTime(&eTime1, start_1, end_1);

	hipEventRecord(start_2);
	hipMemcpy(hO, dO, size, hipMemcpyDeviceToHost);
	hipEventRecord(end_2);
	hipEventSynchronize(end_2);
	float eTime2 = 0;
	hipEventElapsedTime(&eTime2, start_2, end_2);

	hipFree(dA);
	hipFree(dO);
	free(hA);
	free(hO);
	hipEventDestroy(start);
	hipEventDestroy(end);
	hipEventDestroy(start_1);
	hipEventDestroy(end_1);
	hipEventDestroy(start_2);
	hipEventDestroy(end_2);

	printf("%-15d%-15f%-15f%-15f\n",N,eTime,eTime1,eTime2);

	}
}
