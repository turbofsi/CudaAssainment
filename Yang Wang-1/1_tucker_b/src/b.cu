/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 * Yang Wang
 * Department of ECE
 * University of Toronto
 */
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>

__global__ void ADD(float * A, float*O, int N, float x)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < N)
	{
		O[i] = A[i] + x;
	}
}

int main(void)
{
	int 		N = 32 * 1024 * 256;
	double 	input = 32;
	float 		x = 10.0;

	printf("%s\n%s\n%s\n", "FIRSTNAME: Yang", "LASTNAME: Wang",
						"E-MAIL: tucker.wang@mail.utoronto.ca");
	printf("%-15s%-15s%-15s%-15s%-15s\n", "Elements(M)", "Block_size", "CPUtoGPU(ms)",
						"Kernel(ms)", "GPUtoCPU(ms)");

	for(double threadsPerBlock = 16; threadsPerBlock < pow(2.0, 15); threadsPerBlock *= 2)
	{
		size_t 	size = N * sizeof(float);
		float * hA = (float *) malloc(size);
		float * hO = (float *) malloc(size);

		float * dA;
		hipMalloc((void**) &dA, size);
		float * dO;
		hipMalloc((void**) &dO, size);

		hipEvent_t start, start_1, start_2;
		hipEvent_t end, end_1, end_2;

		hipEventCreate(&start);
		hipEventCreate(&start_1);
		hipEventCreate(&start_2);

		hipEventCreate(&end);
		hipEventCreate(&end_1);
		hipEventCreate(&end_2);

		for (int i = 0; i < N; i++) {
			hA[i] = rand() / (float) RAND_MAX;
		}

		hipEventRecord(start);
		hipMemcpy(dA, hA, size, hipMemcpyHostToDevice);
		hipEventRecord(end);
		hipEventSynchronize(end);
		float eTime = 0;
		hipEventElapsedTime(&eTime, start, end);

		int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

		hipEventRecord(start_1);
		ADD<<<blocksPerGrid, threadsPerBlock>>>(dA, dO, N,x);
		hipEventRecord(end_1);
		hipDeviceSynchronize();
		hipEventSynchronize(end_1);
		float eTime1 = 0;
		hipEventElapsedTime(&eTime1, start_1, end_1);

		hipEventRecord(start_2);
		hipMemcpy(hO, dO, size, hipMemcpyDeviceToHost);
		hipEventRecord(end_2);
		hipEventSynchronize(end_2);
		float eTime2 = 0;
		hipEventElapsedTime(&eTime2, start_2, end_2);

		hipFree(dA);
		hipFree(dO);
		free(hA);
		free(hO);
		hipEventDestroy(start);
		hipEventDestroy(end);
		hipEventDestroy(start_1);
		hipEventDestroy(end_1);
		hipEventDestroy(start_2);
		hipEventDestroy(end_2);

		printf("%-15.0f%-15.0f%-15f%-15f%-15f\n", input, threadsPerBlock, eTime,
				eTime1, eTime2);

	}
}
