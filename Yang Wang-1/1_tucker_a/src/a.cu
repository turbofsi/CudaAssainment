/* *
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 * Yang Wang
 * Department of ECE
 * University of Toronto
 */
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>

__global__ void ADD(float * A, float*O, int N,float x)

{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < N)
	{
		O[i] = A[i] + x;
	}

}

int main(void)
{

	float 	x = 10.0;

	printf("%s\n%s\n%s\n", "FIRSTNAME: Yang", "LASTNAME: Wang","E-MAIL: tucker.wang@mail.utoronto.ca");
	printf("%-15s%-15s%-15s%-15s\n", "Elements(M)", "CPUtoGPU(ms)","Kernel(ms)", "GPUtoCPU(ms)");

//Loop Begin
	for(int N = 1024 * 256; N < pow(2.0, 30); N *= 2)
	{
		size_t size = N * sizeof(float);
		float * hA = (float *) malloc(size);
		float * hO = (float *) malloc(size);

		float * dA;
		hipMalloc((void **) &dA, size);
		float * dO;
		hipMalloc((void **) &dO, size);

		hipEvent_t start, start_1, start_2;
		hipEvent_t end, end_1, end_2;

		hipEventCreate(&start);
		hipEventCreate(&start_1);
		hipEventCreate(&start_2);

		hipEventCreate(&end);
		hipEventCreate(&end_1);
		hipEventCreate(&end_2);

		for (int i = 0; i < N; i++)
		{
			hA[i] = rand() / (float) RAND_MAX;
		}

		//CPU2GPU ElapsedTime
		hipEventRecord(start);
		hipMemcpy(dA, hA, size, hipMemcpyHostToDevice);
		hipEventRecord(end);
		hipEventSynchronize(end);
		float eTime = 0;
		hipEventElapsedTime(&eTime, start, end);

		int threadsPerBlock = 256;
		int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

		//KERNEL ELAPSEDTIME
		hipEventRecord(start_1);
		ADD<<<blocksPerGrid, threadsPerBlock>>>(dA, dO, N,x);
		hipEventRecord(end_1);
		hipDeviceSynchronize();
		hipEventSynchronize(end_1);
		float eTime1 = 0;
		hipEventElapsedTime(&eTime1, start_1, end_1);

		//GPU2CPU ELAPSEDTIME
		hipEventRecord(start_2);
		hipMemcpy(hO, dO, size, hipMemcpyDeviceToHost);
		hipEventRecord(end_2);
		hipEventSynchronize(end_2);
		float eTime2 = 0;
		hipEventElapsedTime(&eTime2, start_2, end_2);

		hipFree(dA);
		hipFree(dO);
		free(hA);
		free(hO);

		hipEventDestroy(start);
		hipEventDestroy(start_1);
		hipEventDestroy(start_2);

		hipEventDestroy(end);
		hipEventDestroy(end_1);
		hipEventDestroy(end_2);

		printf("%-15d%-15f%-15f%-15f\n", N / (1024 * 256), eTime, eTime1,
				eTime2);
	}
}



