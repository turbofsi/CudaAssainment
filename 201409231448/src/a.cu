/* *
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>

__global__ void ADD(float * A, float*O, int N) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < N) {
		O[i] = A[i] + threadIdx.x;
	}
}

int main(void) {
	int N = 1024 * 256;
	while (N < pow(2.0, 30)) {
		size_t size = N * sizeof(float);
		float* h_A = (float*) malloc(size);
		float* h_O = (float*) malloc(size);
		float* d_A;
		hipMalloc((void**) &d_A, size);
		float* d_O;
		hipMalloc((void**) &d_O, size);
		hipEvent_t stop, stop1, stop2;
		hipEvent_t start, start1, start2;
		hipEventCreate(&start);
		hipEventCreate(&stop);
		hipEventCreate(&start1);
		hipEventCreate(&stop1);
		hipEventCreate(&start2);
		hipEventCreate(&stop2);
		for (int i = 0; i < N; i++) {
			h_A[i] = rand() / (float) RAND_MAX;
		}
		hipEventRecord(start);
		hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		float elapsedTime = 0;
		hipEventElapsedTime(&elapsedTime, start, stop);
		int threadsPerBlock = 256;
		int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
		hipEventRecord(start1);
		ADD<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_O, N);
		hipEventRecord(stop1);
		hipDeviceSynchronize();
		hipEventSynchronize(stop1);
		float elapsedTime1 = 0;
		hipEventElapsedTime(&elapsedTime1, start1, stop1);
		hipEventRecord(start2);
		hipMemcpy(h_O, d_O, size, hipMemcpyDeviceToHost);
		hipEventRecord(stop2);
		hipEventSynchronize(stop2);
		float elapsedTime2 = 0;
		hipEventElapsedTime(&elapsedTime2, start2, stop2);
		hipFree(d_A);
		hipFree(d_O);
		free(h_A);
		free(h_O);
		hipEventDestroy(start);
		hipEventDestroy(stop);
		hipEventDestroy(start1);
		hipEventDestroy(stop1);
		hipEventDestroy(start2);
		hipEventDestroy(stop2);
		if (N == 1024 *256) {
			printf("%s\n%s\n%s\n", "FIRSTNAME: XINYUN", "LASTNAME: LV",
					"E-MAIL: xinyunlv0425@gmail.com");
			printf("%-28s%-15s%-15s%-15s\n", "Elements(M)", " CPUtoGPU(ms)",
					" Kernel(ms)", " GPUtoCPU(ms)");
		}
		printf("%-30d%-15f%-15f%-15f\n", N / (1024 * 256), elapsedTime, elapsedTime1,
				elapsedTime2);

		N = N * 2;
	}
}
